#include "hip/hip_runtime.h"

/*

   Copyright 2022 Shin Watanabe

   Licensed under the Apache License, Version 2.0 (the "License");
   you may not use this file except in compliance with the License.
   You may obtain a copy of the License at

       http://www.apache.org/licenses/LICENSE-2.0

   Unless required by applicable law or agreed to in writing, software
   distributed under the License is distributed on an "AS IS" BASIS,
   WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
   See the License for the specific language governing permissions and
   limitations under the License.

*/

#include "shared.h"

using namespace shared;

RT_PIPELINE_LAUNCH_PARAMETERS PipelineLaunchParameters1 plp;

CUDA_DEVICE_KERNEL void RT_RG_NAME (rg0)()
{
    uint32_t value = 0;
    optixu::trace<Pipeline0Payload0Signature> (
        plp.travHandle,
        make_float3 (0, 0, 0), make_float3 (0, 0, 1), 0.0f, INFINITY, 0.0f,
        0xFF, OPTIX_RAY_FLAG_NONE,
        0, 1, 0,
        value);
}

CUDA_DEVICE_KERNEL void RT_EX_NAME (ex0)() {}

CUDA_DEVICE_KERNEL void RT_MS_NAME (ms0)() {}

CUDA_DEVICE_KERNEL void RT_CH_NAME (ch0)()
{
    float3 value;
    Pipeline1Payload0Signature::get (&value);
    value = make_float3 (value.x + 1, value.y + 1, value.z + 1);
    Pipeline1Payload0Signature::set (&value);
}

CUDA_DEVICE_KERNEL void RT_CH_NAME (ch1)()
{
    float3 value;
    Pipeline1Payload0Signature::get (&value);
    value = make_float3 (value.x + 2, value.y + 2, value.z + 2);
    Pipeline1Payload0Signature::set (&value);
}

CUDA_DEVICE_KERNEL void RT_AH_NAME (ah0)() {}

CUDA_DEVICE_KERNEL void RT_IS_NAME (is0)() {}

CUDA_DEVICE_KERNEL void RT_DC_NAME (dc0)() {}

CUDA_DEVICE_KERNEL void RT_CC_NAME (cc0)() {}
